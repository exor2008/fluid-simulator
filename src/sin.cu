
#include <hip/hip_runtime.h>
extern "C" __global__ void sin_kernel(float *out, const float *inp, int numel)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numel)
    {
        out[i] = sin(inp[i]);
    }
}