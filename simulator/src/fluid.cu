
#include <hip/hip_runtime.h>
__constant__ int U_FEILD = 0;
__constant__ int V_FEILD = 1;
__constant__ int W_FEILD = 2;
__constant__ int S_FEILD = 3;
__constant__ float FRICTION = 0.8;
__constant__ float H = 1.0 / 100.0;

extern "C" __global__ void divergence(
    float *div,
    const float *u,
    const float *v,
    const float *w,
    const bool *block,
    int x_size,
    int y_size,
    int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        int idx = (y + y_size * z) * x_size + x;

        if (!block[idx])
        {
            float du_dx = u[(y + y_size * z) * x_size + x + 1] - u[idx];
            float dv_dy = v[(y + 1 + y_size * z) * x_size + x] - v[idx];
            float dw_dz = w[(y + y_size * (z + 1)) * x_size + x] - w[idx];
            div[idx] = du_dx + dv_dy + dw_dz;
        }
    }
}

extern "C" __global__ void pressure(
    float *pressure_a,
    float *pressure_b,
    const float *div,
    const bool *block,
    int x_size,
    int y_size,
    int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        int idx = (y + y_size * z) * x_size + x;
        if (!block[idx])
        {
            float sum = (pressure_b[(y + y_size * (z + 1)) * x_size + x] +
                         pressure_b[(y + y_size * (z - 1)) * x_size + x] +
                         pressure_b[(y + 1 + y_size * z) * x_size + x] +
                         pressure_b[(y - 1 + y_size * z) * x_size + x] +
                         pressure_b[(y + y_size * z) * x_size + x + 1] +
                         pressure_b[(y + y_size * z) * x_size + x - 1]);
            pressure_a[idx] = (sum - div[idx]) / 6;
        }
    }
}

extern "C" __global__ void incompress(
    float *u,
    float *v,
    float *w,
    const float *pressure,
    const bool *block,
    int x_size,
    int y_size,
    int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        int idx = (y + y_size * z) * x_size + x;
        if (!block[idx])
        {
            float dp_dx = pressure[(y + y_size * z) * x_size + x + 1] - pressure[idx];
            float dp_dy = pressure[(y + 1 + y_size * z) * x_size + x] - pressure[idx];
            float dp_dz = pressure[(y + y_size * (z + 1)) * x_size + x] - pressure[idx];

            u[idx] -= dp_dx;
            v[idx] -= dp_dy;
            w[idx] -= dp_dz;
        }
    }
}

extern "C" __device__ float avg_u(const float *u, int x_size, int y_size, int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        return (u[(y + y_size * z) * x_size + x] +
                u[(y + y_size * z) * x_size + x + 1] +
                u[(y - 1 + y_size * z) * x_size + x] +
                u[(y - 1 + y_size * z) * x_size + x + 1] +
                u[(y + y_size * (z - 1)) * x_size + x] +
                u[(y + y_size * (z - 1)) * x_size + x + 1] +
                u[(y - 1 + y_size * (z - 1)) * x_size + x] +
                u[(y - 1 + y_size * (z - 1)) * x_size + x + 1]) *
               0.125;
    }
    else
    {
        return 0;
    }
}

extern "C" __device__ float avg_v(const float *v, int x_size, int y_size, int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        return (v[(y + y_size * z) * x_size + x] +
                v[(y + y_size * z) * x_size + x - 1] +
                v[(y + 1 + y_size * z) * x_size + x] +
                v[(y + 1 + y_size * z) * x_size + x - 1] +
                v[(y + y_size * (z - 1)) * x_size + x] +
                v[(y + y_size * (z - 1)) * x_size + x - 1] +
                v[(y + 1 + y_size * (z - 1)) * x_size + x] +
                v[(y + 1 + y_size * (z - 1)) * x_size + x - 1]) *
               0.125;
    }
    else
    {
        return 0;
    }
}

extern "C" __device__ float avg_w(const float *w, int x_size, int y_size, int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        return (w[(y + y_size * z) * x_size + x] +
                w[(y + y_size * z) * x_size + x - 1] +
                w[(y - 1 + y_size * z) * x_size + x] +
                w[(y - 1 + y_size * z) * x_size + x - 1] +
                w[(y + y_size * (z + 1)) * x_size + x] +
                w[(y + y_size * (z + 1)) * x_size + x - 1] +
                w[(y - 1 + y_size * (z + 1)) * x_size + x] +
                w[(y - 1 + y_size * (z + 1)) * x_size + x - 1]) *
               0.125;
    }
    else
    {
        return 0;
    }
}

extern "C" __device__ float sample_field(
    const float *u,
    const float *v, const float *w,
    const float *smoke,
    float x_shift,
    float y_shift,
    float z_shift,
    const int field_type,
    float h,
    int x_size,
    int y_size,
    int z_size)
{
    float h1 = 1.0 / h;
    float h2 = 0.5 * h;

    x_shift = max(min(x_shift, x_size * h), h);
    y_shift = max(min(y_shift, y_size * h), h);
    z_shift = max(min(z_shift, z_size * h), h);

    float dx = 0.0;
    float dy = 0.0;
    float dz = 0.0;
    const float *field;

    if (field_type == U_FEILD)
    {
        field = u;
        dy = h2;
        dz = h2;
    }
    else if (field_type == V_FEILD)
    {
        field = v;
        dx = h2;
        dz = h2;
    }
    else if (field_type == W_FEILD)
    {
        field = w;
        dx = h2;
        dy = h2;
    }
    else
    {
        field = smoke;
        dx = h2;
        dy = h2;
        dz = h2;
    }

    float xs = float(x_size - 1);
    float ys = float(y_size - 1);
    float zs = float(z_size - 1);

    float x0 = min(floor((x_shift - dx) * h1), xs);
    float tx = ((x_shift - dx) - x0 * h) * h1;
    float x1 = min(x0 + 1, xs);

    float y0 = min(floor((y_shift - dy) * h1), ys);
    float ty = ((y_shift - dy) - y0 * h) * h1;
    float y1 = min(y0 + 1, ys);

    float z0 = min(floor((z_shift - dz) * h1), zs);
    float tz = ((z_shift - dz) - z0 * h) * h1;
    float z1 = min(z0 + 1, zs);

    float sx = 1.0 - tx;
    float sy = 1.0 - ty;
    float sz = 1.0 - tz;

    int x_0 = int(x0);
    int x_1 = int(x1);
    int y_0 = int(y0);
    int y_1 = int(y1);
    int z_0 = int(z0);
    int z_1 = int(z1);

    return (
        sy * sz * sx * field[(y_0 + y_size * z_0) * x_size + x_0] +
        sy * sz * tx * field[(y_0 + y_size * z_0) * x_size + x_1] +
        ty * sz * sx * field[(y_1 + y_size * z_0) * x_size + x_0] +
        ty * sz * tx * field[(y_1 + y_size * z_0) * x_size + x_1] +
        sy * tz * sx * field[(y_0 + y_size * z_1) * x_size + x_0] +
        sy * tz * tx * field[(y_0 + y_size * z_1) * x_size + x_1] +
        ty * tz * sx * field[(y_1 + y_size * z_1) * x_size + x_0] +
        ty * tz * tx * field[(y_1 + y_size * z_1) * x_size + x_1]);
}

extern "C" __global__ void advect_velocity(
    const float *u,
    const float *v,
    const float *w,
    float *new_u,
    float *new_v,
    float *new_w,
    const float *smoke,
    const bool *block,
    float dt,
    int x_size,
    int y_size,
    int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float h2 = 0.5 * H;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        int idx = (y + y_size * z) * x_size + x;

        if (!block[idx])
        {
            float avg_u_scalar = avg_u(u, x_size, y_size, z_size);
            float avg_v_scalar = avg_v(v, x_size, y_size, z_size);
            float avg_w_scalar = avg_w(w, x_size, y_size, z_size);

            // U
            float x_shift = x * H;
            float y_shift = y * H + h2;
            float z_shift = z * H + h2;

            float u_scalar = u[idx];
            float v_scalar = avg_v_scalar;
            float w_scalar = avg_w_scalar;

            x_shift -= dt * u_scalar;
            y_shift -= dt * v_scalar;
            z_shift -= dt * w_scalar;

            new_u[idx] = sample_field(u, v, w, smoke, x_shift, y_shift, z_shift, U_FEILD, H, x_size, y_size, z_size);

            // V
            x_shift = x * H + h2;
            y_shift = y * H;
            z_shift = z * H + h2;

            u_scalar = avg_u_scalar;
            v_scalar = v[idx];
            w_scalar = avg_w_scalar;

            x_shift -= dt * u_scalar;
            y_shift -= dt * v_scalar;
            z_shift -= dt * w_scalar;

            new_v[idx] = sample_field(u, v, w, smoke, x_shift, y_shift, z_shift, V_FEILD, H, x_size, y_size, z_size);

            // W
            x_shift = x * H + h2;
            y_shift = y * H + h2;
            z_shift = z * H;

            u_scalar = avg_u_scalar;
            v_scalar = avg_v_scalar;
            w_scalar = w[idx];

            x_shift -= dt * u_scalar;
            y_shift -= dt * v_scalar;
            z_shift -= dt * w_scalar;

            new_w[idx] = sample_field(u, v, w, smoke, x_shift, y_shift, z_shift, W_FEILD, H, x_size, y_size, z_size);
        }
    }
}

extern "C" __global__ void advect_smoke(
    const float *smoke,
    float *new_smoke,
    const float *u,
    const float *v,
    const float *w,
    const bool *block,
    float dt,
    int x_size,
    int y_size,
    int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        float h2 = 0.5 * H;

        int idx = (y + y_size * z) * x_size + x;

        if (!block[idx])
        {
            float u_scalar = (u[idx] + u[(y + y_size * z) * x_size + x + 1]) * 0.5;
            float v_scalar = (v[idx] + v[(y + 1 + y_size * z) * x_size + x]) * 0.5;
            float w_scalar = (w[idx] + w[(y + y_size * (z + 1)) * x_size + x]) * 0.5;

            float x_shift = x * H + h2 - dt * u_scalar;
            float y_shift = y * H + h2 - dt * v_scalar;
            float z_shift = z * H + h2 - dt * w_scalar;

            float val = sample_field(u, v, w, smoke, x_shift, y_shift, z_shift, S_FEILD, H, x_size, y_size, z_size);
            new_smoke[idx] = val;
        }
    }
}

extern "C" __device__ void flip_vector(
    float x,
    float y,
    float z,
    float normal_x,
    float normal_y,
    float normal_z,
    float &flipped_x,
    float &flipped_y,
    float &flipped_z)
{
    float dot = x * normal_x + y * normal_y + z * normal_z;

    flipped_x = x - 2 * dot * normal_x;
    flipped_y = y - 2 * dot * normal_y;
    flipped_z = z - 2 * dot * normal_z;
}

extern "C" __global__ void calc_borders(
    float *u,
    float *v,
    float *w,
    const float *normal_u,
    const float *normal_v,
    const float *normal_w,
    const bool *block,
    int x_size,
    int y_size,
    int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    float flipped_x;
    float flipped_y;
    float flipped_z;

    if (x > 0 && y > 0 && z > 0 && x < x_size - 1 && y < y_size - 1 && z < z_size - 1)
    {
        int idx = (y + y_size * z) * x_size + x;

        if (!block[idx])
        {
            if (normal_u[idx] != 0 || normal_v[idx] != 0 || normal_w[idx] != 0)
            {
                flip_vector(u[idx], v[idx], w[idx], normal_u[idx], normal_v[idx], normal_w[idx], flipped_x, flipped_y, flipped_z);

                u[idx] = flipped_x * FRICTION;
                v[idx] = flipped_y * FRICTION;
                w[idx] = flipped_z * FRICTION;
            }
        }
    }
}

extern "C" __global__ void constant(
    float *u,
    float *w,
    float *smoke,
    const bool *block,
    int x_size,
    int y_size,
    int z_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= 20 && x < 30 && y >= 45 && y < y_size - 45 && z >= 35 && z < z_size - 35)
    {
        int idx = (y + y_size * z) * x_size + x;

        if (!block[idx])
        {
            u[idx] = 3.0;
            smoke[idx] = 1.0;
        }
    }
}